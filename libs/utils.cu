#include <stdio.h>
#include <stdlib.h>
#include "utils.h"
#include "graph.h"

double gettime( void )
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

// Import graph from a file
struct Graph *importGraphFromFile(const char *filename) {
    FILE *file = fopen(filename, "r");
	int V, E;
    if (file != NULL) {
		fscanf(file, "%d", &V);
		fscanf(file, "%d", &E);
		struct Graph* graph = createGraph(V, E);
		int count = 0;
		int tmp;
        for (int i = 0; i < V; i++) {
            for (int j = 0; j < V; j++) {
                fscanf(file, "%d", &tmp);
				if(tmp != 0){
					// printf("Edge %d: %d %d %d\n", count, i, j, tmp);
					graph->edge[count].src = i;
					graph->edge[count].dest = j;
					graph->edge[count].weight = tmp;
					count++;
				}
			}
    	}
		fclose(file);
		return graph;
	} else {
		printf("Error opening file for reading.\n");
		return NULL;
    }
    return NULL;
}

// Free device memory
void freeDevice(struct Graph* d_graph, int* d_dist, int* d_isNegativeCycle) {
    hipFree(d_isNegativeCycle); // Free isNegativeCycle
    hipFree(d_dist); // Free distances
    hipFree(&(d_graph->edge)); // Free edges
    hipFree(d_graph); // Free graph
}

// Free host memory
void freeHost(struct Graph* h_graph, int* h_dist) {
    free(h_graph->edge); // Free edges
    free(h_graph); // Free graph
    free(h_dist); // Free distances
}