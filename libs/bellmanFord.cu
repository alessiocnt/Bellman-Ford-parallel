#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bellmanFord.h"
#include "graph.h"

__global__ void bellmanFord(Graph *d_graph, int *d_dist) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_graph->E) {
        int u = d_graph->edge[i].src;
        int v = d_graph->edge[i].dest;
        int weight = d_graph->edge[i].weight;

        if (d_dist[u] != INT_MAX && d_dist[u] + weight < d_dist[v]) {
            atomicMin(&d_dist[v], d_dist[u] + weight);
        }
    }
}

__global__ void checkNegativeWeightCycles(Graph *d_graph, int *d_dist, int *d_isNegativeCycle) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_graph->E) {
        int u = d_graph->edge[i].src;
        int v = d_graph->edge[i].dest;
        int weight = d_graph->edge[i].weight;

        if (d_dist[u] != INT_MAX && d_dist[u] + weight < d_dist[v]) {
            d_dist[v] = -INT_MAX;
            *d_isNegativeCycle = 1;
        }
    }
}