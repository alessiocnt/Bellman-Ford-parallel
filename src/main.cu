#include <stdio.h>
#include <stdlib.h>
#include "bellmanFord.h"
#include "graph.h"
#include "utils.h"

int main()
{
    printf("MAIN\n");
	struct Graph *h_graph, *d_graph;
    int h_isNegativeCycle = 0;
    int *h_dist, *d_dist, *d_isNegativeCycle;
    // Host structures
    printf("Create graph...\n");
    h_graph = importGraphFromFile("./graph_test.txt");
    int V = h_graph->V;
    int E = h_graph->E;
    printf("Graph imported\n");

    h_dist = (int*)malloc(V * sizeof(int));
    // Initialize distances from src to all other vertices as infinite
    for (int i=0; i<V; i++)
        h_dist[i] = INT_MAX;
    h_dist[0] = 0;

    // Device structures
    // Graph structure
    hipMalloc((void**)&d_graph, sizeof(struct Graph));
    hipMemcpy(d_graph, h_graph, sizeof(struct Graph), hipMemcpyHostToDevice);
    Edge *d_edge;
    hipMalloc((void**)&d_edge, E*sizeof(struct Edge));
    hipMemcpy(d_edge, h_graph->edge, E*sizeof(struct Edge), hipMemcpyHostToDevice);
    hipMemcpy(&(d_graph->edge), &d_edge, sizeof(Edge*), hipMemcpyHostToDevice);
    // Distances
    hipMalloc((void **)&d_dist, V*sizeof(int));
    hipMemcpy(d_dist, h_dist, V*sizeof(int), hipMemcpyHostToDevice);
    // Is negative cycle
    hipMalloc((void**)&d_isNegativeCycle, sizeof(int));
    hipMemcpy(d_isNegativeCycle, &h_isNegativeCycle, sizeof(int), hipMemcpyHostToDevice);
	
    // Define grid and block dimensions
    int blockSize = 256;
    int numBlocks = (E + blockSize - 1) / blockSize;

    double tstart = gettime();
    // Run kernel V-1 times
    for (int i=0; i<(V-1); i++) {
        bellmanFord<<<numBlocks, blockSize>>>(d_graph, d_dist);
        hipDeviceSynchronize(); // Ensure all threads finish before next iteration
    }
    // Run kernel to check for negative-weight cycles
    checkNegativeWeightCycles<<<numBlocks, blockSize>>>(d_graph, d_dist, d_isNegativeCycle);
    hipDeviceSynchronize(); // Ensure all threads finish before copying result back to host
    double tend = gettime();
    printf("Elapsed time in seconds for parallel implementation: %f\n", tend-tstart);
    
    // Copy result back to host
    hipMemcpy(&h_isNegativeCycle, d_isNegativeCycle, sizeof(int), hipMemcpyDeviceToHost);
    if (h_isNegativeCycle) {
        printf("Graph contains negative weight cycle\n");
        return 0;
    } 
    hipMemcpy(h_dist, d_dist, V * sizeof(int), hipMemcpyDeviceToHost);
    // Print distances
    // printf("Vertex   Distance from Source\n");
    // for (int i = 0; i < V; i++) {
    //     printf("%d \t\t %d\n", i, h_dist[i]);
    // }

    freeHost(h_graph, h_dist);
    freeDevice(d_graph, d_dist, d_isNegativeCycle);
    return 0;
}