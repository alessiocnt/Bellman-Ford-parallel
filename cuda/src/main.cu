#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>

#define ITERATIONS 5

double gettime( void )
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts );
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

// A structure to represent a weighted edge in graph
struct Edge {
	int src, dest, weight;
};
// A structure to represent a connected, directed and weighted graph
struct Graph {
    int V, E;
    struct Edge* edge;
};
// Creates a graph
// V: number of vertices, E: number of edges
struct Graph* createGraph(int V, int E) {
    struct Graph* graph = (struct Graph*)malloc(sizeof(struct Graph));
    graph->V = V;
    graph->E = E;
    graph->edge = (struct Edge*)malloc(E * sizeof(struct Edge));
    return graph;
}

char* getFilename(char *filename) {
	char *path = "./benchmark/";
	char *fullPath = (char*)malloc(strlen(path) + strlen(filename) + 1);
	strcpy(fullPath, path);
	strcat(fullPath, filename);
	return fullPath;
}

// Import graph from a file
struct Graph *importGraphFromFile(const char *filename) {
    FILE *file = fopen(filename, "r");
	int V, E;
    if (file != NULL) {
		fscanf(file, "%d", &V);
		fscanf(file, "%d", &E);
		struct Graph* graph = createGraph(V, E);
		int count = 0;
		int tmp;
        for (int i = 0; i < V; i++) {
            for (int j = 0; j < V; j++) {
                fscanf(file, "%d", &tmp);
				if(tmp != 0){
					// printf("Edge %d: %d %d %d\n", count, i, j, tmp);
					graph->edge[count].src = i;
					graph->edge[count].dest = j;
					graph->edge[count].weight = tmp;
					count++;
				}
			}
    	}
		fclose(file);
		return graph;
	} else {
		printf("Error opening file for reading.\n");
		return NULL;
    }
    return NULL;
}

// Free device memory
void freeTemporaryDevice(int* d_dist, int* d_isNegativeCycle) {
    hipFree(d_isNegativeCycle); // Free isNegativeCycle
    hipFree(d_dist); // Free distances
}

void freeGraphDevice(struct Graph* d_graph) {
    hipFree(&(d_graph->edge)); // Free edges
    hipFree(d_graph); // Free graph
}

// Free host memory
void freeHost(struct Graph* h_graph) {
    free(h_graph->edge); // Free edges
    free(h_graph); // Free graph
}

void freeTemporaryHost(int* h_dist) {
    free(h_dist); // Free distances
}

__global__ void bellmanFord(Graph *d_graph, int *d_dist) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_graph->E) {
        int u = d_graph->edge[i].src;
        int v = d_graph->edge[i].dest;
        int weight = d_graph->edge[i].weight;

        if (d_dist[u] != INT_MAX && d_dist[u] + weight < d_dist[v]) {
            atomicMin(&d_dist[v], d_dist[u] + weight);
        }
    }
}

__global__ void checkNegativeWeightCycles(Graph *d_graph, int *d_dist, int *d_isNegativeCycle) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < d_graph->E) {
        int u = d_graph->edge[i].src;
        int v = d_graph->edge[i].dest;
        int weight = d_graph->edge[i].weight;

        if (d_dist[u] != INT_MAX && d_dist[u] + weight < d_dist[v]) {
            d_dist[v] = -INT_MAX;
            *d_isNegativeCycle = 1;
        }
    }
}

int main(int argc, char* argv[]) {
    char** file_names;
	if (argc == 2) {
		file_names = (char**)malloc(2 * sizeof(const char*));
		file_names[0] = argv[1];
		file_names[1] = NULL; // Mark the end of the array
	} else {
		file_names = (char**)malloc(5 * sizeof(const char*));
        file_names[0] = "graph_XS.txt";
        file_names[1] = "graph_S.txt";
	file_names[2] = "graph_M.txt";
	file_names[3] = "graph_L.txt";
        file_names[4] = NULL;
	}
	printf("\n ---- CUDA ----\n");
    // Loop through each file name
    int file_number = 0;
	char *path;
    while (file_names[file_number] != NULL) {
		path = getFilename(file_names[file_number]);
        struct Graph *h_graph, *d_graph;
        // Host graph structures
        printf("Importing %s ...\n", file_names[file_number]);
        h_graph = importGraphFromFile(path);
        int V = h_graph->V;
        int E = h_graph->E;
        printf("Graph imported.\n");
        // Device graph structures
        hipMalloc((void**)&d_graph, sizeof(struct Graph));
        hipMemcpy(d_graph, h_graph, sizeof(struct Graph), hipMemcpyHostToDevice);
        Edge *d_edge;
        hipMalloc((void**)&d_edge, E*sizeof(struct Edge));
        hipMemcpy(d_edge, h_graph->edge, E*sizeof(struct Edge), hipMemcpyHostToDevice);
        hipMemcpy(&(d_graph->edge), &d_edge, sizeof(Edge*), hipMemcpyHostToDevice);

        double timing[ITERATIONS];
		double totalTime = 0;

        for (int i=0; i<ITERATIONS; i++) {
            // Host initialization
            int h_isNegativeCycle = 0;
            int *h_dist, *d_dist, *d_isNegativeCycle;
            h_dist = (int*)malloc(V * sizeof(int));
            // Initialize distances from src to all other vertices as infinite
            for (int j=0; j<V; j++)
                h_dist[j] = INT_MAX;
            h_dist[0] = 0;
            // Device initialization
            // Distances
            hipMalloc((void **)&d_dist, V*sizeof(int));
            hipMemcpy(d_dist, h_dist, V*sizeof(int), hipMemcpyHostToDevice);
            // Is negative cycle
            hipMalloc((void**)&d_isNegativeCycle, sizeof(int));
            hipMemcpy(d_isNegativeCycle, &h_isNegativeCycle, sizeof(int), hipMemcpyHostToDevice);

            // Define grid and block dimensions
            int blockSize = 256;
            int numBlocks = (E + blockSize - 1) / blockSize;
            double tstart = gettime();
            // Run kernel V-1 times
            for (int j=0; j<(V-1); j++) {
                bellmanFord<<<numBlocks, blockSize>>>(d_graph, d_dist);
                hipDeviceSynchronize(); // Ensure all threads finish before next iteration
            }
            // Run kernel to check for negative-weight cycles
            checkNegativeWeightCycles<<<numBlocks, blockSize>>>(d_graph, d_dist, d_isNegativeCycle);
            hipDeviceSynchronize(); // Ensure all threads finish before copying result back to host
            double tend = gettime();
            double elapsed = tend - tstart; 
            // printf("Elapsed time in seconds for parallel implementation: %f\n", elapsed);
            timing[i] = elapsed;
            
            // Copy result back to host
            hipMemcpy(&h_isNegativeCycle, d_isNegativeCycle, sizeof(int), hipMemcpyDeviceToHost);
            if (h_isNegativeCycle) {
                printf("Graph contains negative weight cycle\n");
                return 0;
            } 
            hipMemcpy(h_dist, d_dist, V * sizeof(int), hipMemcpyDeviceToHost);
            // Print distances
            // printf("Vertex   Distance from Source\n");
            // for (int i = 0; i < V; i++) {
            //     printf("%d \t\t %d\n", i, h_dist[i]);
            // }
            freeTemporaryDevice(d_dist, d_isNegativeCycle);
            freeTemporaryHost(h_dist);
        }
        printf("Time for execution: ");
        for (int j = 0; j<ITERATIONS; j++) {
            printf("%f ", timing[j]);
            totalTime += timing[j];
        }
        printf("\n Average time: %f\n", totalTime/ITERATIONS);
        printf("\n ---------------------- \n");
        totalTime = 0;
        file_number++;
        freeHost(h_graph);
        freeGraphDevice(d_graph);   
    }

    return EXIT_SUCCESS;
}
